#include "hip/hip_runtime.h"
#include <omp.h>
#include "vecscatter.h"
#include "brick.h"


// $START naive
__global__ void f3d_naive(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0], bElem (*c)[8][8]) {
    const size_t radius = $SIZE;
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;

    bElem base = 0;
    #pragma unroll
    for (int i_diff = -radius; i_diff <= radius; i_diff++) {
        #pragma unroll
        for (int j_diff = -radius; j_diff <= radius; j_diff++) {
            #pragma unroll
            for (int k_diff = -radius; k_diff <= radius; k_diff++) {
                base += (in[i + i_diff][j + j_diff][k + k_diff] * c[i_diff + radius][j_diff + radius][k_diff + radius]);
            }
        }
    }
    out[i][j][k] = base;
}
// $END naive

// $START naive-bricks
__global__ void f3d_naive_bricks(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem (*c)[8][8]) {
    const size_t radius = $SIZE;
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;

    bElem base = 0;
    #pragma unroll
    for (int i_diff = -radius; i_diff <= radius; i_diff++) {
        #pragma unroll
        for (int j_diff = -radius; j_diff <= radius; j_diff++) {
            #pragma unroll
            for (int k_diff = -radius; k_diff <= radius; k_diff++) {
                base += (bIn[b][i + i_diff][j + j_diff][k + k_diff] * c[i_diff + radius][j_diff + radius][k_diff + radius]);
            }
        }
    }
    bOut[b][i][j][k] = base;
}
// $END naive-bricks

// $START codegen
#define bIn(a, b, c) arr_in[c][b][a]
#define bOut(a, b, c) arr_out[c][b][a]

__global__ void f3d_codegen(bElem (*arr_in)[STRIDE1][STRIDE0], bElem (*arr_out)[STRIDE1][STRIDE0], bElem (*c)[8][8]) {
    long k = OFF2 + (blockIdx.z * TILE2);
    long j = OFF1 + (blockIdx.y * TILE1);
    long i = OFF0 + (blockIdx.x * VECSIZE);
    tile("$PYTHON", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}

#undef bIn
#undef bOut
// $END codegen

// $START codegen-bricks
__global__ void f3d_codegen_bricks(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType bIn, BType bOut, bElem (*c)[8][8]) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    brick("$PYTHON", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
// $END codegen-bricks
